#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NB 32
#define NT 500
#define N NB*NT

__global__ void add( double *a, double *b, double *c);

//===========================================
__global__ void add( double *a, double *b, double *c){

    int tid = threadIdx.x + blockIdx.x * blockDim.x; 

    while(tid < N){
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }

}

//============================================
//BEGIN
//===========================================
int main( void ) {

    double *a, *b, *c;
    double *dev_a, *dev_b, *dev_c;

    // allocate the memory on the CPU
    a=(double *)malloc(N*sizeof(double));
    b=(double *)malloc(N*sizeof(double));
    c=(double *)malloc(N*sizeof(double));

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(double) );
    hipMalloc( (void**)&dev_b, N * sizeof(double) );
    hipMalloc( (void**)&dev_c, N * sizeof(double) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = (double)i;
        b[i] = (double)i*2;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy( dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);

    for(int i=0;i<10000;++i)
        add<<<NB,NT>>>( dev_a, dev_b, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c, dev_c, N * sizeof(double), hipMemcpyDeviceToHost);

    // display the results
    // for (int i=0; i<N; i++) {
    //      printf( "%g + %g = %g\n", a[i], b[i], c[i] );
    //  }
    printf("\nGPU done\n");

    // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    // free the memory allocated on the CPU
    free( a );
    free( b );
    free( c );

    return 0;
}
